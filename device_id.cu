
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA devices found.\n");
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        printf("Device %d: %s\n", i, deviceProp.name);
    }

    return 0;
}

