#include "hip/hip_runtime.h"
// Stress Cache line - 
// 1 thread writes others just copy data into scratchpad(shared) memory
// Ideas from https://www.cis.upenn.edu/~devietti/classes/cis601-spring2017/slides/gpu-concurrency.pdf

#include <cuda/atomic>
#include <cstdio>
#include <iostream>
#include <cstdlib> // for rand() and srand()
#include <ctime>   // for time()

using namespace cuda;
#define THREADS 1024


__global__ void consumer(atomic<int>* flag, int* data, atomic<int>* umflag_near_flag, int *um_buffer1, int* result0/*flag*/, int*result1/*data*/) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid == 0) {   // some random thread
        *result0 = flag->load(memory_order_relaxed);
        *result1 = *data;    
    }// else{
    //         // for (int i=0; i< 4096; i++)
    //             um_buffer1[tid] = umflag_near_flag->load(memory_order_relaxed);
    // }
}

int all_are_same(const int *result) {
    int sum = 0;
    for (int i = 0; i < THREADS; ++i) {
        sum += result[i];
    }
    float avg = (float)sum / THREADS; // Convert sum to float before division
    if (avg != result[0]) {
        // for (int i = 0; i < THREADS; ++i) {
        //     printf("%d - ", result[i]);
        // }
        // printf("\n Avg = %f", avg);
        // printf("\n Result[0] = %d", result[0]);
        return 0;
    } else {
        return 1;
    }
}
    
#define SAFE(x) if (0 != x) { abort(); }

void caching(atomic<int> *x, int *y, atomic<int> *flag, int *data ){

    for (int i=0; i<100; i++){
        *x = *data;
        *y = flag->load(memory_order_relaxed);
    }
}
struct Result{
int seq1, seq2, interleave, weak;
};


void run(Result *count_local){
    // int THREADS = 1024;

    atomic<int>* flag;
    atomic<int>* umflag_near_flag;
    int* data;
    int* um_buffer1;  // page size bytes
    int* result0;
    int* result1; // r0= flag, r1=data, GPUHarbor way

    int data_in_unified_memory = 1;

    ////////////////////////////////////////////////////////////////////////////

    // Flag in unified memory
    SAFE(hipMallocManaged(&flag, sizeof(atomic<int>)));
    SAFE(hipMallocManaged(&umflag_near_flag, sizeof(atomic<int>)));

    // Data placed as specified
    if (data_in_unified_memory) {
        SAFE(hipMallocManaged(&data, sizeof(int)));
        SAFE(hipMallocManaged(&um_buffer1, 4096 * sizeof(int)));
    } else {
        SAFE(hipMalloc(&data, sizeof(int)));
        SAFE(hipMalloc(&um_buffer1, 4096 * sizeof(int))); 
    }

    // Result array pinned in CPU memory
    SAFE(hipHostMalloc(&result0, sizeof(int)));
    SAFE(hipHostMalloc(&result1, sizeof(int)));

    // Initial values: data = <unknown>, flag = 0
    flag->store(0, memory_order_relaxed);   // real one
    // umflag_near_flag->store(100, memory_order_relaxed); // fake 
    *data = 0;  // real one
    // random values
    // srand(static_cast<unsigned int>(time(nullptr)));
    // for (int i=0; i< 4096; i++)
    //     um_buffer1[i] = rand() %100 + 1;  // fake
    

    ////////////////////////////////////////////////////////////////////////////

    // Launch the consumer asynchronously
    // consumer<<<1,1>>>(flag, data, result0, result1);
    
    // gpU
    // max go till 2 blocks and n threads.
    consumer<<<1,1024>>>(flag, data, umflag_near_flag, um_buffer1, result0, result1); 

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(error));
        exit(1);
    }
    
    // // THIS IS THE PART WHICH TRIGGERS LOTS OF BEHAVIORS
    // // Read data and cache it in CPU.
    // for (int i=0; i< 1024; i++){
    //     um_buffer1[i] = *data;    // reads data and cache it in CPU side.
    //     umflag_near_flag->store(flag->load(memory_order_relaxed), memory_order_relaxed);    // cross read to make more confusion.
    // }
    // cpu allocation data doesn't help
    // 


    atomic<int> *x;//= (int*)malloc(sizeof(int));
    int *y;// = (int*)malloc(sizeof(int));
    SAFE(hipMallocManaged(&x, sizeof(atomic<int>)));   // x == flag
    SAFE(hipMallocManaged(&y, sizeof(int)));   // y==data
    caching(x,y,flag, data);
    // Producer sequence
    if (data_in_unified_memory) {
        
        *data = 42;
        // // for no reason write in this buffer
        // for (int i=0; i< 4096; i++)
        //     um_buffer1[i] = rand() %100 + 1;  // fake        
    } else {
        int h_data = 42;
        SAFE(hipMemcpy(data, &h_data, sizeof(int), hipMemcpyHostToDevice));
    }
    // for (int i=0; i<100; i++){
    //     *x = *data;
    //     *y = flag->load(memory_order_relaxed);
    // }    
    flag->store(1, memory_order_relaxed);


  // Wait for consumer to finish
    SAFE(hipDeviceSynchronize());

    // // perform some checks to make sure all are same
    // if ((all_are_same(result0) == 0) || (all_are_same(result1) == 0)){  // someone is not same
    //     printf("\n Isssue with implementation please fix!");
    // }


     //r0=flag, r1=data
    if (*result0 == 0 && *result1 == 0){
        count_local->seq1 += 1 ;  //# t1->t2
    }
    else if(*result0 == 1 && *result1 == 42){
        count_local->seq2 += 1 ;  //# t2-t1
    }
    else if(*result0 == 0 && *result1 == 42){
        count_local->interleave += 1;
    }
    else if(*result0 == 1 && *result1 == 0){
        count_local->weak += 1;
    }
        
       // Free the allocated memory at the end
    SAFE(hipFree(flag));
    if (data_in_unified_memory) {
        SAFE(hipFree(data));
    } else {
        SAFE(hipFree(data));
    }
    SAFE(hipHostFree(result0));
    SAFE(hipHostFree(result1));


}
int main(int argc, char* argv[]) {
        if (argc !=2 ){
            printf("\n ./a.out <number of tests>");
            exit(1);
        }
     int loop_size = atoi(argv[1]);
     Result count_local{0};

    for (int i=0; i< loop_size; i++){
        // printf("i=%d\n", i);
        run(&count_local);
        if (i == loop_size/4){
            printf("\n 25%%");
        } else if (i == loop_size/2){
            printf("\n 50%%");
        }
    }

    printf("\n Histogram after %d runs\n", loop_size);
    printf("seq1 (flag)=0; (data)=0;  = %d\n", count_local.seq1);
    printf("seq2 (flag)=1; (data)=42; = %d\n", count_local.seq2);
    printf("intlv(flag)=0; (data)=42; = %d\n", count_local.interleave);
    printf("weak (flag)=1; (data)=0;  = %d\n", count_local.weak);

    return 0;
}

