
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  int d;
  hipGetDevice(&d);

  int pma = 0;
  hipDeviceGetAttribute(&pma, hipDeviceAttributePageableMemoryAccess, d);
  printf("cudaDevAttrPageableMemoryAccess: %s\n", pma == 1? "YES" : "NO");
  
  int cma = 0;
  hipDeviceGetAttribute(&cma, hipDeviceAttributeConcurrentManagedAccess, d);
  printf("cudaDevAttrConcurrentManagedAccess: %s\n", cma == 1? "YES" : "NO");


  int mm = 0;
  hipDeviceGetAttribute(&mm, hipDeviceAttributeManagedMemory, d);
  printf("managedMemory: %s\n", mm == 1? "YES" : "NO");


  int dma = 0;
  hipDeviceGetAttribute(&dma, hipDeviceAttributeDirectManagedMemAccessFromHost, d);
  printf("directManagedMemAccessFromHost: %s\n", dma == 1? "YES" : "NO");


  int x = 0;
  hipDeviceGetAttribute(&x, hipDeviceAttributePageableMemoryAccessUsesHostPageTables, d);
  printf("pageableMemoryAccessUsesHostPageTables: %s\n", x == 1? "YES" : "NO");


  return 0;
}
