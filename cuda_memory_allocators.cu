
#include <hip/hip_runtime.h>
#include <stdio.h>

char const* kind(hipPointerAttribute_t a, bool pma, bool cma) {
    switch(a.type) {
    case hipMemoryTypeHost: return pma?
      "Unified: CUDA Host or Registered Memory" :
      "Not Unified: CUDA Host or Registered Memory";
    case hipMemoryTypeDevice: return "Not Unified: CUDA Device Memory";
    case hipMemoryTypeManaged: return cma?
      "Unified: CUDA Managed Memory" : "Not Unified: CUDA Managed Memory";
    case cudaMemoryTypeUnregistered: return pma?
      "Unified: System-Allocated Memory" :
      "Not Unified: System-Allocated Memory";
    default: return "unknown";
    }
}

void check_pointer(int i, void* ptr) {
  hipPointerAttribute_t attr;
  hipPointerGetAttributes(&attr, ptr);
  int pma = 0, cma = 0, device = 0;
  hipGetDevice(&device);
  hipDeviceGetAttribute(&pma, hipDeviceAttributePageableMemoryAccess, device);
  hipDeviceGetAttribute(&cma, hipDeviceAttributeConcurrentManagedAccess, device);
  printf("Pointer %d: memory is %s\n", i, kind(attr, pma, cma));
}

__managed__ int managed_var = 5;

int main() {
  int* ptr[5];
  ptr[0] = (int*)malloc(sizeof(int));
  hipMallocManaged(&ptr[1], sizeof(int));
  hipHostMalloc(&ptr[2], sizeof(int), hipHostMallocDefault);
  hipMalloc(&ptr[3], sizeof(int));
  ptr[4] = &managed_var;

  for (int i = 0; i < 5; ++i) check_pointer(i, ptr[i]);
  
  hipFree(ptr[3]);
  hipHostFree(ptr[2]);
  hipFree(ptr[1]);
  free(ptr[0]);
  return 0;
}
